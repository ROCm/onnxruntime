#include "hip/hip_runtime.h"
/*
 The implementation of this file is based on skipLayerNorm plugin in TensorRT demo:
 https://github.com/NVIDIA/TensorRT/tree/release/5.1/demo/BERT/
 
Copyright 2019 NVIDIA Corporation

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License.
*/

// Copyright (c) Microsoft Corporation. All rights reserved.
// Licensed under the MIT License.

#include "core/providers/rocm/bert/layer_norm.cuh"
#include "contrib_ops/rocm/bert/skip_layer_norm_impl.h"
#include <hip/hip_fp16.h>

namespace onnxruntime {
namespace contrib {
namespace rocm {

template <typename T, unsigned TPB>
__global__ void SkipLayerNormKernel(
    const int ld, const T* input, const T* skip, const T* beta, const T* gamma,
    const T epsilon, T* output) {
  const T reverse_ld = T(1.f / ld);
  const int offset = blockIdx.x * ld;
  extern __shared__ unsigned char shared_memory_[];
  T* shared_memory = reinterpret_cast<T*>(shared_memory_);

  KeyValuePairSum pair_sum;
  // reduce x and x^2
  hipcub::KeyValuePair<T, T> thread_data(0, 0);
  const int idx = offset + threadIdx.x;
  T val = 0;

  // if (threadIdx.x >= TPB) return;
  shared_memory[threadIdx.x] = input[idx];
  shared_memory[threadIdx.x + TPB] = skip[idx];

  __syncthreads();

  if (threadIdx.x < ld) {
    val = shared_memory[threadIdx.x] + shared_memory[threadIdx.x + TPB];
    const T rldval = reverse_ld * val;
    thread_data = pair_sum(thread_data, hipcub::KeyValuePair<T, T>(rldval, rldval * val));
  }

  LayerNormSmall<T, TPB>(val, thread_data, ld, idx, beta, gamma, epsilon, output);
}

// separate -bias and -nobias
// put input and skip into "__shared__"
template <typename T, unsigned TPB>
__global__ void SkipLayerNormKernelBias(
    const int ld, const T* input, const T* skip, const T* beta, const T* gamma, const T* bias,
    const T epsilon, T* output) {
  const T reverse_ld = T(1.f / ld);
  const int offset = blockIdx.x * ld;
  extern __shared__ unsigned char shared_memory_[];
  T* shared_memory = reinterpret_cast<T*>(shared_memory_);

  KeyValuePairSum pair_sum;
  // reduce x and x^2
  hipcub::KeyValuePair<T, T> thread_data(0, 0);
  const int idx = offset + threadIdx.x;
  T val = 0;

  // if (threadIdx.x >= TPB) return; // redundant
  shared_memory[threadIdx.x] = input[idx];
  shared_memory[threadIdx.x + TPB] = skip[idx];
  shared_memory[threadIdx.x + 2 * TPB] = bias[threadIdx.x];
  __syncthreads();

  if (threadIdx.x < ld) {
    val = shared_memory[threadIdx.x] + shared_memory[threadIdx.x + TPB] + shared_memory[threadIdx.x + 2 * TPB];
    const T rldval = reverse_ld * val;
    thread_data = pair_sum(thread_data, hipcub::KeyValuePair<T, T>(rldval, rldval * val));
  }

  LayerNormSmall<T, TPB>(val, thread_data, ld, idx, beta, gamma, epsilon, output);
}

template <typename T>
bool ComputeSkipLayerNorm(
    hipStream_t stream, const int ld, const int n, const T* input, const T* skip,
    const T* beta, const T* gamma, const T* bias, const T epsilon, T* output) {
  // this must be true because n is the total size of the tensor
  assert(n % ld == 0);
  const int grid_size = n / ld;  // hidden_size: ld,  element_count: n;

  if (ld <= 32) {
    constexpr int block_size = 32;
    if (bias == nullptr) {
      unsigned int shmem = block_size * sizeof(T) * 2;
      //SkipLayerNormKernel<T, block_size>
      //  <<<grid_size, block_size, shmem, stream>>>(ld, input, skip, beta, gamma, epsilon, output);
      hipLaunchKernelGGL(HIP_KERNEL_NAME(SkipLayerNormKernel<T, block_size>), dim3(grid_size), dim3(block_size), smem,
                     stream, ld, input, skip, beta, gamma, epsilon, output);
    } else {
      unsigned int shmem = block_size * sizeof(T) * 3;
      //SkipLayerNormKernelBias<T, block_size>
      //  <<<grid_size, block_size, shmem, stream>>>(ld, input, skip, beta, gamma, bias, epsilon, output);
      hipLaunchKernelGGL(HIP_KERNEL_NAME(SkipLayerNormKernelBias<T, block_size>), dim3(grid_size), dim3(block_size), smem,
                     stream, ld, input, skip, beta, gamma, bias, epsilon, output);
    }
  } else if (ld <= 128) {
    constexpr int block_size = 128;
    if (bias == nullptr) {
      unsigned int shmem = block_size * sizeof(T) * 2;
      //SkipLayerNormKernel<T, block_size>
      //  <<<grid_size, block_size, shmem, stream>>>(ld, input, skip, beta, gamma, epsilon, output);
      hipLaunchKernelGGL(HIP_KERNEL_NAME(SkipLayerNormKernel<T, block_size>), dim3(grid_size), dim3(block_size), smem,
                     stream, ld, input, skip, beta, gamma, epsilon, output);
    } else {
      unsigned int shmem = block_size * sizeof(T) * 3;
      //SkipLayerNormKernelBias<T, block_size>
      //  <<<grid_size, block_size, shmem, stream>>>(ld, input, skip, beta, gamma, bias, epsilon, output);
      hipLaunchKernelGGL(HIP_KERNEL_NAME(SkipLayerNormKernelBias<T, block_size>), dim3(grid_size), dim3(block_size), smem,
                     stream, ld, input, skip, beta, gamma, bias, epsilon, output);
    }
  } else if (ld == 384) {
    constexpr int block_size = 384;
    if (bias == nullptr) {
      unsigned int shmem = block_size * sizeof(T) * 2;
      //SkipLayerNormKernel<T, block_size>
      //  <<<grid_size, block_size, shmem, stream>>>(ld, input, skip, beta, gamma, epsilon, output);
      hipLaunchKernelGGL(HIP_KERNEL_NAME(SkipLayerNormKernel<T, block_size>), dim3(grid_size), dim3(block_size), smem,
                     stream, ld, input, skip, beta, gamma, epsilon, output);
    } else {
      unsigned int shmem = block_size * sizeof(T) * 3;
      //SkipLayerNormKernelBias<T, block_size>
      //  <<<grid_size, block_size, shmem, stream>>>(ld, input, skip, beta, gamma, bias, epsilon, output);
      hipLaunchKernelGGL(HIP_KERNEL_NAME(SkipLayerNormKernelBias<T, block_size>), dim3(grid_size), dim3(block_size), smem,
                     stream, ld, input, skip, beta, gamma, bias, epsilon, output);
    }
  } else {
    constexpr int block_size = 256;
    if (bias == nullptr) {
      unsigned int shmem = block_size * sizeof(T) * 2;
      //SkipLayerNormKernel<T, block_size>
      //  <<<grid_size, block_size, shmem, stream>>>(ld, input, skip, beta, gamma, epsilon, output);
      hipLaunchKernelGGL(HIP_KERNEL_NAME(SkipLayerNormKernel<T, block_size>), dim3(grid_size), dim3(block_size), smem,
                     stream, ld, input, skip, beta, gamma, epsilon, output);
    } else {
      unsigned int shmem = block_size * sizeof(T) * 3;
      //SkipLayerNormKernelBias<T, block_size>
      //  <<<grid_size, block_size, shmem, stream>>>(ld, input, skip, beta, gamma, bias, epsilon, output);
      hipLaunchKernelGGL(HIP_KERNEL_NAME(SkipLayerNormKernelBias<T, block_size>), dim3(grid_size), dim3(block_size), smem,
                     stream, ld, input, skip, beta, gamma, bias, epsilon, output);
    }
  }
  //return CUDA_CALL(hipPeekAtLastError());
  return HIP_CALL(hipPeekAtLastError());
}

bool LaunchSkipLayerNormKernel(
    hipStream_t stream,
    void* output,
    const void* input,
    const void* skip,
    const void* gamma,
    const void* beta,
    const void* bias,
    float epsilon,
    int hidden_size,
    int element_count,
    size_t element_size) {
  if (element_size == 2) {
    return ComputeSkipLayerNorm(
        stream,
        hidden_size,
        element_count,
        reinterpret_cast<const half*>(input),
        reinterpret_cast<const half*>(skip),
        reinterpret_cast<const half*>(beta),
        reinterpret_cast<const half*>(gamma),
        reinterpret_cast<const half*>(bias),
        __float2half_rn(epsilon),
        reinterpret_cast<half*>(output));
  } else {
    return ComputeSkipLayerNorm(
        stream,
        hidden_size,
        element_count,
        reinterpret_cast<const float*>(input),
        reinterpret_cast<const float*>(skip),
        reinterpret_cast<const float*>(beta),
        reinterpret_cast<const float*>(gamma),
        reinterpret_cast<const float*>(bias),
        epsilon,
        reinterpret_cast<float*>(output));
  }
}

}  // namespace rocm
}  // namespace contrib
}  // namespace onnxruntime
